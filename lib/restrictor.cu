#include "hip/hip_runtime.h"
#include <color_spinor_field.h>
#include <color_spinor_field_order.h>
#include <tune_quda.h>
#include <hipcub/hipcub.hpp>
#include <typeinfo>
#include <multigrid_helper.cuh>
#include <fast_intdiv.h>

// enabling CTA swizzling improves spatial locality of MG blocks reducing cache line wastage
#ifndef SWIZZLE
#define SWIZZLE
#endif

namespace quda {

#ifdef GPU_MULTIGRID

  using namespace quda::colorspinor;

  /** 
      Kernel argument struct
  */
  template <typename Float, typename vFloat, int fineSpin, int fineColor,
	    int coarseSpin, int coarseColor, QudaFieldOrder order>
  struct RestrictArg {

    FieldOrderCB<Float,coarseSpin,coarseColor,1,order> out;
    const FieldOrderCB<Float,fineSpin,fineColor,1,order> in;
    const FieldOrderCB<Float,fineSpin,fineColor,coarseColor,order,vFloat> V;
    const int *fine_to_coarse;
    const int *coarse_to_fine;
    const spin_mapper<fineSpin,coarseSpin> spin_map;
    const int parity; // the parity of the input field (if single parity)
    const int nParity; // number of parities of input fine field
    int_fastdiv swizzle; // swizzle factor for transposing blockIdx.x mapping to coarse grid coordinate

    RestrictArg(ColorSpinorField &out, const ColorSpinorField &in, const ColorSpinorField &V,
		const int *fine_to_coarse, const int *coarse_to_fine, int parity)
      : out(out), in(in), V(V), fine_to_coarse(fine_to_coarse), coarse_to_fine(coarse_to_fine),
	spin_map(), parity(parity), nParity(in.SiteSubset()), swizzle(1)
    { }

    RestrictArg(const RestrictArg<Float,vFloat,fineSpin,fineColor,coarseSpin,coarseColor,order> &arg) :
      out(arg.out), in(arg.in), V(arg.V), 
      fine_to_coarse(arg.fine_to_coarse), coarse_to_fine(arg.coarse_to_fine), spin_map(),
      parity(arg.parity), nParity(arg.nParity), swizzle(arg.swizzle)
    { }
  };

  /**
     Rotates from the fine-color basis into the coarse-color basis.
  */
  template <typename Float, int fineSpin, int fineColor, int coarseColor, int coarse_colors_per_thread,
	    class FineColor, class Rotator>
  __device__ __host__ inline void rotateCoarseColor(complex<Float> out[fineSpin*coarse_colors_per_thread],
						    const FineColor &in, const Rotator &V,
						    int parity, int nParity, int x_cb, int coarse_color_block) {
    const int spinor_parity = (nParity == 2) ? parity : 0;
    const int v_parity = (V.Nparity() == 2) ? parity : 0;

#pragma unroll
    for (int s=0; s<fineSpin; s++)
#pragma unroll
      for (int coarse_color_local=0; coarse_color_local<coarse_colors_per_thread; coarse_color_local++) {
	out[s*coarse_colors_per_thread+coarse_color_local] = 0.0;
      }

#pragma unroll
    for (int coarse_color_local=0; coarse_color_local<coarse_colors_per_thread; coarse_color_local++) {
      int i = coarse_color_block + coarse_color_local;
#pragma unroll
      for (int s=0; s<fineSpin; s++) {

	constexpr int color_unroll = fineColor == 3 ? 3 : 2;

	complex<Float> partial[color_unroll];
#pragma unroll
	for (int k=0; k<color_unroll; k++) partial[k] = 0.0;

#pragma unroll
	for (int j=0; j<fineColor; j+=color_unroll) {
#pragma unroll
	  for (int k=0; k<color_unroll; k++)
	    partial[k] += conj(V(v_parity, x_cb, s, j+k, i)) * in(spinor_parity, x_cb, s, j+k);
	}

#pragma unroll
	for (int k=0; k<color_unroll; k++) out[s*coarse_colors_per_thread + coarse_color_local] += partial[k];
      }
    }

  }

  template <typename Float, int fineSpin, int fineColor, int coarseSpin, int coarseColor, int coarse_colors_per_thread, typename Arg>
  void Restrict(Arg arg) {
    for (int parity_coarse=0; parity_coarse<2; parity_coarse++) 
      for (int x_coarse_cb=0; x_coarse_cb<arg.out.VolumeCB(); x_coarse_cb++)
	for (int s=0; s<coarseSpin; s++) 
	  for (int c=0; c<coarseColor; c++)
	    arg.out(parity_coarse, x_coarse_cb, s, c) = 0.0;

    // loop over fine degrees of freedom
    for (int parity=0; parity<arg.nParity; parity++) {
      parity = (arg.nParity == 2) ? parity : arg.parity;

      for (int x_cb=0; x_cb<arg.in.VolumeCB(); x_cb++) {

	int x = parity*arg.in.VolumeCB() + x_cb;
	int x_coarse = arg.fine_to_coarse[x];
	int parity_coarse = (x_coarse >= arg.out.VolumeCB()) ? 1 : 0;
	int x_coarse_cb = x_coarse - parity_coarse*arg.out.VolumeCB();
	
	for (int coarse_color_block=0; coarse_color_block<coarseColor; coarse_color_block+=coarse_colors_per_thread) {
	  complex<Float> tmp[fineSpin*coarse_colors_per_thread];
	  rotateCoarseColor<Float,fineSpin,fineColor,coarseColor,coarse_colors_per_thread>
	    (tmp, arg.in, arg.V, parity, arg.nParity, x_cb, coarse_color_block);

	  for (int s=0; s<fineSpin; s++) {
	    for (int coarse_color_local=0; coarse_color_local<coarse_colors_per_thread; coarse_color_local++) {
	      int c = coarse_color_block + coarse_color_local;
	      arg.out(parity_coarse,x_coarse_cb,arg.spin_map(s,parity),c) += tmp[s*coarse_colors_per_thread+coarse_color_local];
	    }
	  }

	}
      }
    }

  }

  /**
     struct which acts as a wrapper to a vector of data.
   */
  template <typename scalar, int n>
  struct vector_type {
    scalar data[n];
    __device__ __host__ inline scalar& operator[](int i) { return data[i]; }
    __device__ __host__ inline const scalar& operator[](int i) const { return data[i]; }
    __device__ __host__ inline static constexpr int size() { return n; }
    __device__ __host__ vector_type() { for (int i=0; i<n; i++) data[i] = 0.0; }
  };

  /**
     functor that defines how to do a multi-vector reduction
   */
  template <typename T>
  struct reduce {
    __device__ __host__ inline T operator()(const T &a, const T &b) {
      T sum;
      for (int i=0; i<sum.size(); i++) sum[i] = a[i] + b[i];
      return sum;
    }
  };

  /**
     Here, we ensure that each thread block maps exactly to a
     geometric block.  Each thread block corresponds to one geometric
     block, with number of threads equal to the number of fine grid
     points per aggregate, so each thread represents a fine-grid
     point.  The look up table coarse_to_fine is the mapping to
     each fine grid point.
  */
  template <typename Float, int fineSpin, int fineColor, int coarseSpin, int coarseColor, int coarse_colors_per_thread,
	    typename Arg, int block_size>
  __global__ void RestrictKernel(Arg arg) {

#ifdef SWIZZLE
    // the portion of the grid that is exactly divisible by the number of SMs
    const int gridp = gridDim.x - gridDim.x % arg.swizzle;

    int x_coarse = blockIdx.x;
    if (blockIdx.x < gridp) {
      // this is the portion of the block that we are going to transpose
      const int i = blockIdx.x % arg.swizzle;
      const int j = blockIdx.x / arg.swizzle;

      // tranpose the coordinates
      x_coarse = i * (gridp / arg.swizzle) + j;
    }
#else
    int x_coarse = blockIdx.x;
#endif

    int parity_coarse = x_coarse >= arg.out.VolumeCB() ? 1 : 0;
    int x_coarse_cb = x_coarse - parity_coarse*arg.out.VolumeCB();

    // obtain fine index from this look up table
    // since both parities map to the same block, each thread block must do both parities

    // threadIdx.x - fine checkboard offset
    // threadIdx.y - fine parity offset
    // blockIdx.x  - which coarse block are we working on (swizzled to improve cache efficiency)
    // assume that coarse_to_fine look up map is ordered as (coarse-block-id + fine-point-id)
    // and that fine-point-id is parity ordered
    int parity = arg.nParity == 2 ? threadIdx.y : arg.parity;
    int x_fine = arg.coarse_to_fine[ (x_coarse*2 + parity) * blockDim.x + threadIdx.x];
    int x_fine_cb = x_fine - parity*arg.in.VolumeCB();

    int coarse_color_block = (blockDim.z*blockIdx.z + threadIdx.z) * coarse_colors_per_thread;
    if (coarse_color_block >= coarseColor) return;

    complex<Float> tmp[fineSpin*coarse_colors_per_thread];
    rotateCoarseColor<Float,fineSpin,fineColor,coarseColor,coarse_colors_per_thread>
      (tmp, arg.in, arg.V, parity, arg.nParity, x_fine_cb, coarse_color_block);

    typedef vector_type<complex<Float>, coarseSpin*coarse_colors_per_thread> vector;
    vector reduced;

    // first lets coarsen spin locally
    for (int s=0; s<fineSpin; s++) {
      for (int v=0; v<coarse_colors_per_thread; v++) {
	reduced[arg.spin_map(s,parity)*coarse_colors_per_thread+v] += tmp[s*coarse_colors_per_thread+v];
      }
    }

    // now lets coarsen geometry across threads
    if (arg.nParity == 2) {
      typedef hipcub::BlockReduce<vector, block_size, hipcub::BLOCK_REDUCE_WARP_REDUCTIONS, 2> BlockReduce;
      __shared__ typename BlockReduce::TempStorage temp_storage;
      reduce<vector> reducer; // reduce functor

      // note this is not safe for blockDim.z > 1
      reduced = BlockReduce(temp_storage).Reduce(reduced, reducer);
    } else {
      typedef hipcub::BlockReduce<vector, block_size, hipcub::BLOCK_REDUCE_WARP_REDUCTIONS> BlockReduce;
      __shared__ typename BlockReduce::TempStorage temp_storage;
      reduce<vector> reducer; // reduce functor

      // note this is not safe for blockDim.z > 1
      reduced = BlockReduce(temp_storage).Reduce(reduced, reducer);
    }

    if (threadIdx.x==0 && threadIdx.y == 0) {
      for (int s=0; s<coarseSpin; s++) {
	for (int coarse_color_local=0; coarse_color_local<coarse_colors_per_thread; coarse_color_local++) {
	  int v = coarse_color_block + coarse_color_local;
	  arg.out(parity_coarse, x_coarse_cb, s, v) = reduced[s*coarse_colors_per_thread+coarse_color_local];
	}
      }
    }
  }

  template <typename Float, typename vFloat, int fineSpin, int fineColor, int coarseSpin, int coarseColor,
	    int coarse_colors_per_thread>
  class RestrictLaunch : public Tunable {

  protected:
    ColorSpinorField &out;
    const ColorSpinorField &in;
    const ColorSpinorField &v;
    const int *fine_to_coarse;
    const int *coarse_to_fine;
    const int parity;
    const QudaFieldLocation location;
    const int block_size;
    char vol[TuneKey::volume_n];

    unsigned int sharedBytesPerThread() const { return 0; }
    unsigned int sharedBytesPerBlock(const TuneParam &param) const { return 0; }
    bool tuneGridDim() const { return false; } // Don't tune the grid dimensions.
    bool tuneAuxDim() const { return true; } // Do tune the aux dimensions.
    unsigned int minThreads() const { return in.VolumeCB(); } // fine parity is the block y dimension

  public:
    RestrictLaunch(ColorSpinorField &out, const ColorSpinorField &in, const ColorSpinorField &v,
		   const int *fine_to_coarse, const int *coarse_to_fine, int parity)
      : out(out), in(in), v(v), fine_to_coarse(fine_to_coarse), coarse_to_fine(coarse_to_fine),
	parity(parity), location(checkLocation(out,in,v)), block_size(in.VolumeCB()/(2*out.VolumeCB()))
    {
      strcpy(vol, out.VolString());
      strcat(vol, ",");
      strcat(vol, in.VolString());

      strcpy(aux, out.AuxString());
      strcat(aux, ",");
      strcat(aux, in.AuxString());
    } // block size is checkerboard fine length / full coarse length
    virtual ~RestrictLaunch() { }

    void apply(const hipStream_t &stream) {
      if (location == QUDA_CPU_FIELD_LOCATION) {
	if (out.FieldOrder() == QUDA_SPACE_SPIN_COLOR_FIELD_ORDER) {
	  RestrictArg<Float,vFloat,fineSpin,fineColor,coarseSpin,coarseColor,QUDA_SPACE_SPIN_COLOR_FIELD_ORDER>
	    arg(out, in, v, fine_to_coarse, coarse_to_fine, parity);
	  Restrict<Float,fineSpin,fineColor,coarseSpin,coarseColor,coarse_colors_per_thread>(arg);
	} else {
	  errorQuda("Unsupported field order %d", out.FieldOrder());
	}
      } else {
	TuneParam tp = tuneLaunch(*this, getTuning(), getVerbosity());

	if (out.FieldOrder() == QUDA_FLOAT2_FIELD_ORDER) {
	  typedef RestrictArg<Float,vFloat,fineSpin,fineColor,coarseSpin,coarseColor,QUDA_FLOAT2_FIELD_ORDER> Arg;
	  Arg arg(out, in, v, fine_to_coarse, coarse_to_fine, parity);
	  arg.swizzle = tp.aux.x;

	  if (block_size == 4) {          // for 2x2x2x1 aggregates
	    RestrictKernel<Float,fineSpin,fineColor,coarseSpin,coarseColor,coarse_colors_per_thread,Arg,4>
	      <<<tp.grid, tp.block, tp.shared_bytes, stream>>>(arg);
	  } else if (block_size == 2) {   // for 2x2x1x1 aggregates
	    RestrictKernel<Float,fineSpin,fineColor,coarseSpin,coarseColor,coarse_colors_per_thread,Arg,8>
	      <<<tp.grid, tp.block, tp.shared_bytes, stream>>>(arg);
	  } else if (block_size == 8) {   // for 2x2x2x2 aggregates
	    RestrictKernel<Float,fineSpin,fineColor,coarseSpin,coarseColor,coarse_colors_per_thread,Arg,8>
	      <<<tp.grid, tp.block, tp.shared_bytes, stream>>>(arg);
    } else if (block_size == 12) {  // for 2x2x2x3 aggregates
	    RestrictKernel<Float,fineSpin,fineColor,coarseSpin,coarseColor,coarse_colors_per_thread,Arg,12>
	      <<<tp.grid, tp.block, tp.shared_bytes, stream>>>(arg);
    } else if (block_size == 16) {  // for 4x2x2x2 aggregates
	    RestrictKernel<Float,fineSpin,fineColor,coarseSpin,coarseColor,coarse_colors_per_thread,Arg,16>
	      <<<tp.grid, tp.block, tp.shared_bytes, stream>>>(arg);
    } else if (block_size == 18) {  // for 2x2x3x3 aggregates
	    RestrictKernel<Float,fineSpin,fineColor,coarseSpin,coarseColor,coarse_colors_per_thread,Arg,18>
	      <<<tp.grid, tp.block, tp.shared_bytes, stream>>>(arg);
	  } else if (block_size == 27) {  // for 3x3x3x2 aggregates
	    RestrictKernel<Float,fineSpin,fineColor,coarseSpin,coarseColor,coarse_colors_per_thread,Arg,27>
	      <<<tp.grid, tp.block, tp.shared_bytes, stream>>>(arg);
	  } else if (block_size == 32) {  // for 4x4x2x2 aggregates
	    RestrictKernel<Float,fineSpin,fineColor,coarseSpin,coarseColor,coarse_colors_per_thread,Arg,32>
	      <<<tp.grid, tp.block, tp.shared_bytes, stream>>>(arg);
	  } else if (block_size == 36) {  // for 3x3x2x4 aggregates
	    RestrictKernel<Float,fineSpin,fineColor,coarseSpin,coarseColor,coarse_colors_per_thread,Arg,36>
	      <<<tp.grid, tp.block, tp.shared_bytes, stream>>>(arg);
	  } else if (block_size == 40) {  // for 3x3x3x3 aggregates
	    RestrictKernel<Float,fineSpin,fineColor,coarseSpin,coarseColor,coarse_colors_per_thread,Arg,40>
	      <<<tp.grid, tp.block, tp.shared_bytes, stream>>>(arg);
	  } else if (block_size == 54) {  // for 3x3x3x4 aggregates
	    RestrictKernel<Float,fineSpin,fineColor,coarseSpin,coarseColor,coarse_colors_per_thread,Arg,54>
	      <<<tp.grid, tp.block, tp.shared_bytes, stream>>>(arg);
	  } else if (block_size == 64) {  // for 2x4x4x4 aggregates
	    RestrictKernel<Float,fineSpin,fineColor,coarseSpin,coarseColor,coarse_colors_per_thread,Arg,64>
	      <<<tp.grid, tp.block, tp.shared_bytes, stream>>>(arg);
	  } else if (block_size == 81) {  // for 3x3x3x6 aggregates
	    RestrictKernel<Float,fineSpin,fineColor,coarseSpin,coarseColor,coarse_colors_per_thread,Arg,81>
	      <<<tp.grid, tp.block, tp.shared_bytes, stream>>>(arg);
	  } else if (block_size == 96) {  // for 4x4x3x4 aggregates
	    RestrictKernel<Float,fineSpin,fineColor,coarseSpin,coarseColor,coarse_colors_per_thread,Arg,96>
	      <<<tp.grid, tp.block, tp.shared_bytes, stream>>>(arg);
	  } else if (block_size == 72) {  // for 4x4x3x3 aggregates
	    RestrictKernel<Float,fineSpin,fineColor,coarseSpin,coarseColor,coarse_colors_per_thread,Arg,96>
	      <<<tp.grid, tp.block, tp.shared_bytes, stream>>>(arg);
	  } else if (block_size == 100) {  // for 5x5x2x4 aggregates
	    RestrictKernel<Float,fineSpin,fineColor,coarseSpin,coarseColor,coarse_colors_per_thread,Arg,100>
	      <<<tp.grid, tp.block, tp.shared_bytes, stream>>>(arg);
	  } else if (block_size == 108) {  // for 3x3x3x8 aggregates
	    RestrictKernel<Float,fineSpin,fineColor,coarseSpin,coarseColor,coarse_colors_per_thread,Arg,108>
	      <<<tp.grid, tp.block, tp.shared_bytes, stream>>>(arg);
	  } else if (block_size == 128) { // for 4x4x4x4 aggregates
	    RestrictKernel<Float,fineSpin,fineColor,coarseSpin,coarseColor,coarse_colors_per_thread,Arg,128>
	      <<<tp.grid, tp.block, tp.shared_bytes, stream>>>(arg);
	  } else if (block_size == 144) {  // for 4x4x3x6 aggregates
	    RestrictKernel<Float,fineSpin,fineColor,coarseSpin,coarseColor,coarse_colors_per_thread,Arg,144>
	      <<<tp.grid, tp.block, tp.shared_bytes, stream>>>(arg);
#if __COMPUTE_CAPABILITY__ >= 300
	  } else if (block_size == 192) {  // for 4x4x3x8 aggregates
	    RestrictKernel<Float,fineSpin,fineColor,coarseSpin,coarseColor,coarse_colors_per_thread,Arg,192>
	      <<<tp.grid, tp.block, tp.shared_bytes, stream>>>(arg);
	  } else if (block_size == 200) { // for 5x5x2x8  aggregates
	    RestrictKernel<Float,fineSpin,fineColor,coarseSpin,coarseColor,coarse_colors_per_thread,Arg,200>
	      <<<tp.grid, tp.block, tp.shared_bytes, stream>>>(arg);
	  } else if (block_size == 256) { // for 4x4x4x8  aggregates
	    RestrictKernel<Float,fineSpin,fineColor,coarseSpin,coarseColor,coarse_colors_per_thread,Arg,256>
	      <<<tp.grid, tp.block, tp.shared_bytes, stream>>>(arg);
	  } else if (block_size == 288) { // for 4x4x6x6  aggregates
	    RestrictKernel<Float,fineSpin,fineColor,coarseSpin,coarseColor,coarse_colors_per_thread,Arg,288>
	      <<<tp.grid, tp.block, tp.shared_bytes, stream>>>(arg);
	  } else if (block_size == 432) { // for 6x6x6x4 aggregates
	    RestrictKernel<Float,fineSpin,fineColor,coarseSpin,coarseColor,coarse_colors_per_thread,Arg,432>
	      <<<tp.grid, tp.block, tp.shared_bytes, stream>>>(arg);
	  } else if (block_size == 500) { // 5x5x5x8 aggregates
	    RestrictKernel<Float,fineSpin,fineColor,coarseSpin,coarseColor,coarse_colors_per_thread,Arg,500>
	      <<<tp.grid, tp.block, tp.shared_bytes, stream>>>(arg);
	  } else if (block_size == 512) { // 4x4x8x8 aggregates
	    RestrictKernel<Float,fineSpin,fineColor,coarseSpin,coarseColor,coarse_colors_per_thread,Arg,512>
	      <<<tp.grid, tp.block, tp.shared_bytes, stream>>>(arg);
#endif
	  } else {
	    errorQuda("Block size %d not instantiated", block_size);
	  }
	} else {
	  errorQuda("Unsupported field order %d", out.FieldOrder());
	}
      }
    }

    // This block tuning tunes for the optimal amount of color
    // splitting between blockDim.z and gridDim.z.  However, enabling
    // blockDim.z > 1 gives incorrect results due to cub reductions
    // being unable to do independent sliced reductions along
    // blockDim.z.  So for now we only split between colors per thread
    // and grid.z.
    bool advanceBlockDim(TuneParam &param) const
    {
      // let's try to advance spin/block-color
      while(param.block.z <= coarseColor/coarse_colors_per_thread) {
	param.block.z++;
	if ( (coarseColor/coarse_colors_per_thread) % param.block.z == 0) {
	  param.grid.z = (coarseColor/coarse_colors_per_thread) / param.block.z;
	  break;
	}
      }

      // we can advance spin/block-color since this is valid
      if (param.block.z <= (coarseColor/coarse_colors_per_thread) ) { //
	return true;
      } else { // we have run off the end so let's reset
	param.block.z = 1;
	param.grid.z = coarseColor/coarse_colors_per_thread;
	return false;
      }
    }

    int tuningIter() const { return 3; }

    bool advanceAux(TuneParam &param) const
    {
#ifdef SWIZZLE
      if (param.aux.x < 2*deviceProp.multiProcessorCount) {
        param.aux.x++;
	return true;
      } else {
        param.aux.x = 1;
	return false;
      }
#else
      return false;
#endif
    }

    // only tune shared memory per thread (disable tuning for block.z for now)
    bool advanceTuneParam(TuneParam &param) const { return advanceSharedBytes(param) || advanceAux(param); }

    TuneKey tuneKey() const { return TuneKey(vol, typeid(*this).name(), aux); }

    void initTuneParam(TuneParam &param) const { defaultTuneParam(param); }

    /** sets default values for when tuning is disabled */
    void defaultTuneParam(TuneParam &param) const {
      param.block = dim3(block_size, in.SiteSubset(), 1);
      param.grid = dim3( (minThreads()+param.block.x-1) / param.block.x, 1, 1);
      param.shared_bytes = 0;

      param.block.z = 1;
      param.grid.z = coarseColor / coarse_colors_per_thread;
      param.aux.x = 1; // swizzle factor
    }

    long long flops() const { return 8 * fineSpin * fineColor * coarseColor * in.SiteSubset()*(long long)in.VolumeCB(); }

    long long bytes() const {
      size_t v_bytes = v.Bytes() / (v.SiteSubset() == in.SiteSubset() ? 1 : 2);
      return in.Bytes() + out.Bytes() + v_bytes + in.SiteSubset()*in.VolumeCB()*sizeof(int);
    }

  };

  template <typename Float, int fineSpin, int fineColor, int coarseSpin, int coarseColor>
  void Restrict(ColorSpinorField &out, const ColorSpinorField &in, const ColorSpinorField &v,
		const int *fine_to_coarse, const int *coarse_to_fine, int parity) {

    // for fine grids (Nc=3) have more parallelism so can use more coarse strategy
    constexpr int coarse_colors_per_thread = fineColor != 3 ? 2 : coarseColor >= 4 && coarseColor % 4 == 0 ? 4 : 2;
    //coarseColor >= 8 && coarseColor % 8 == 0 ? 8 : coarseColor >= 4 && coarseColor % 4 == 0 ? 4 : 2;

    if (v.Precision() == QUDA_HALF_PRECISION) {
      RestrictLaunch<Float, short, fineSpin, fineColor, coarseSpin, coarseColor, coarse_colors_per_thread>
	restrictor(out, in, v, fine_to_coarse, coarse_to_fine, parity);
      restrictor.apply(0);
    } else if (v.Precision() == in.Precision()) {
      RestrictLaunch<Float, Float, fineSpin, fineColor, coarseSpin, coarseColor, coarse_colors_per_thread>
	restrictor(out, in, v, fine_to_coarse, coarse_to_fine, parity);
      restrictor.apply(0);
    } else {
      errorQuda("Unsupported V precision %d", v.Precision());
    }

    if (checkLocation(out, in, v) == QUDA_CUDA_FIELD_LOCATION) checkCudaError();
  }

  template <typename Float, int fineSpin>
  void Restrict(ColorSpinorField &out, const ColorSpinorField &in, const ColorSpinorField &v,
		int nVec, const int *fine_to_coarse, const int *coarse_to_fine, const int * const * spin_map, int parity) {

    if (out.Nspin() != 2) errorQuda("Unsupported nSpin %d", out.Nspin());
    const int coarseSpin = 2;

    // first check that the spin_map matches the spin_mapper
    spin_mapper<fineSpin,coarseSpin> mapper;
    for (int s=0; s<fineSpin; s++) 
      for (int p=0; p<2; p++)
        if (mapper(s,p) != spin_map[s][p]) errorQuda("Spin map does not match spin_mapper");


    // Template over fine color
    if (in.Ncolor() == 3) { // standard QCD
      const int fineColor = 3;
      if (nVec == 4) {
	Restrict<Float,fineSpin,fineColor,coarseSpin,4>(out, in, v, fine_to_coarse, coarse_to_fine, parity);
      } else if (nVec == 6) { // free field Wilson
  Restrict<Float,fineSpin,fineColor,coarseSpin,6>(out, in, v, fine_to_coarse, coarse_to_fine, parity);
      } else if (nVec == 24) {
	Restrict<Float,fineSpin,fineColor,coarseSpin,24>(out, in, v, fine_to_coarse, coarse_to_fine, parity);
      } else if (nVec == 32) {
	Restrict<Float,fineSpin,fineColor,coarseSpin,32>(out, in, v, fine_to_coarse, coarse_to_fine, parity);
      } else {
	errorQuda("Unsupported nVec %d", nVec);
      }
    } else if (in.Ncolor() == 6) { // Coarsen coarsened Wilson free field
      const int fineColor = 6;
      if (nVec == 6) { 
  Restrict<Float,fineSpin,fineColor,coarseSpin,6>(out, in, v, fine_to_coarse, coarse_to_fine, parity);
      } else {
  errorQuda("Unsupported nVec %d", nVec);
      }
    } else if (in.Ncolor() == 24) { // to keep compilation under control coarse grids have same or more colors
      const int fineColor = 24;
      if (nVec == 24) {
	Restrict<Float,fineSpin,fineColor,coarseSpin,24>(out, in, v, fine_to_coarse, coarse_to_fine, parity);
      } else if (nVec == 32) {
	Restrict<Float,fineSpin,fineColor,coarseSpin,32>(out, in, v, fine_to_coarse, coarse_to_fine, parity);
      } else {
	errorQuda("Unsupported nVec %d", nVec);
      }
    } else if (in.Ncolor() == 32) {
      const int fineColor = 32;
      if (nVec == 32) {
	Restrict<Float,fineSpin,fineColor,coarseSpin,32>(out, in, v, fine_to_coarse, coarse_to_fine, parity);
      } else {
	errorQuda("Unsupported nVec %d", nVec);
      }
    } else {
      errorQuda("Unsupported nColor %d", in.Ncolor());
    }
  }

  template <typename Float>
  void Restrict(ColorSpinorField &out, const ColorSpinorField &in, const ColorSpinorField &v,
		int Nvec, const int *fine_to_coarse, const int *coarse_to_fine, const int * const * spin_map, int parity) {

    if (in.Nspin() == 2) {
      Restrict<Float,2>(out, in, v, Nvec, fine_to_coarse, coarse_to_fine, spin_map, parity);
#ifdef GPU_WILSON_DIRAC
    } else if (in.Nspin() == 4) {
      Restrict<Float,4>(out, in, v, Nvec, fine_to_coarse, coarse_to_fine, spin_map, parity);
#endif
#if GPU_STAGGERED_DIRAC
    } else if (in.Nspin() == 1) {
      Restrict<Float,1>(out, in, v, Nvec, fine_to_coarse, coarse_to_fine, spin_map, parity);
#endif
    } else {
      errorQuda("Unsupported nSpin %d", in.Nspin());
    }
  }

#endif // GPU_MULTIGRID

  void Restrict(ColorSpinorField &out, const ColorSpinorField &in, const ColorSpinorField &v,
		int Nvec, const int *fine_to_coarse, const int *coarse_to_fine, const int * const * spin_map, int parity) {

#ifdef GPU_MULTIGRID
    if (out.FieldOrder() != in.FieldOrder() ||	out.FieldOrder() != v.FieldOrder())
      errorQuda("Field orders do not match (out=%d, in=%d, v=%d)",
		out.FieldOrder(), in.FieldOrder(), v.FieldOrder());

    QudaPrecision precision = checkPrecision(out, in);

    if (precision == QUDA_DOUBLE_PRECISION) {
#ifdef GPU_MULTIGRID_DOUBLE
      Restrict<double>(out, in, v, Nvec, fine_to_coarse, coarse_to_fine, spin_map, parity);
#else
      errorQuda("Double precision multigrid has not been enabled");
#endif
    } else if (precision == QUDA_SINGLE_PRECISION) {
      Restrict<float>(out, in, v, Nvec, fine_to_coarse, coarse_to_fine, spin_map, parity);
    } else {
      errorQuda("Unsupported precision %d", out.Precision());
    }
#else
    errorQuda("Multigrid has not been built");
#endif
  }

} // namespace quda
